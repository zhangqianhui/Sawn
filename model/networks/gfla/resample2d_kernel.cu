#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Context.h>
#include <ATen/cuda/HIPContext.h>

#define CUDA_NUM_THREADS 256 
#define THREADS_PER_BLOCK 64 

#define DIM0(TENSOR) ((TENSOR).x)
#define DIM1(TENSOR) ((TENSOR).y)
#define DIM2(TENSOR) ((TENSOR).z)
#define DIM3(TENSOR) ((TENSOR).w)

#define DIM3_INDEX(TENSOR, xx, yy, zz, ww) ((TENSOR)[((xx) * (TENSOR##_stride.x)) + ((yy) * (TENSOR##_stride.y)) + ((zz) * (TENSOR##_stride.z)) + ((ww) * (TENSOR##_stride.w))])
#define EPS 1e-8
#define SAFE_DIV(a, b)  ( (b==0)? ( (a)/(EPS) ): ( (a)/(b) )  )




template <typename scalar_t>
__global__ void kernel_resample2d_update_output(const int n, 
                                               const scalar_t* __restrict__ input1, const long4 input1_size, const long4 input1_stride,
                                               const scalar_t* __restrict__ input2, const long4 input2_size, const long4 input2_stride, 
                                               scalar_t* __restrict__ output, const long4 output_size, const long4 output_stride, int kernel_size, int dilation) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n) {
        return;
    }

    scalar_t val = 0.0f;
    scalar_t sum = 0.0f;


    int dim_b = DIM0(output_size);
    int dim_c = DIM1(output_size);
    int dim_h = DIM2(output_size);
    int dim_w = DIM3(output_size);
    int dim_chw = dim_c * dim_h * dim_w;
    int dim_hw  = dim_h * dim_w;

    int b = ( index / dim_chw ) % dim_b;
    int c = ( index / dim_hw )  % dim_c;
    int y = ( index / dim_w )   % dim_h;
    int x = ( index          )  % dim_w;

    scalar_t dx    = DIM3_INDEX(input2, b, 0, y, x);
    scalar_t dy    = DIM3_INDEX(input2, b, 1, y, x);
    scalar_t sigma = DIM3_INDEX(input2, b, 2, y, x);


    scalar_t xf = static_cast<scalar_t>(x) + dx;
    scalar_t yf = static_cast<scalar_t>(y) + dy;
    scalar_t alpha = xf - floor(xf); // alpha
    scalar_t beta = yf - floor(yf); // beta


    int idim_h = DIM2(input1_size);
    int idim_w = DIM3(input1_size);


    for (int fy = 0; fy < kernel_size/2; fy += 1) {
        int yT = max(min( int (floor(yf)-fy*dilation),    idim_h-1), 0);
        int yB = max(min( int (floor(yf)+(fy+1)*dilation),idim_h-1), 0);

        for (int fx = 0; fx < kernel_size/2; fx += 1) {
            int xL = max(min( int (floor(xf)-fx*dilation  ),    idim_w-1), 0);
            int xR = max(min( int (floor(xf)+(fx+1)*dilation),  idim_w-1), 0);

            scalar_t xL_ = ( static_cast<scalar_t>( fx    *dilation)+alpha );
            scalar_t xR_ = ( static_cast<scalar_t>((1.+fx)*dilation)-alpha );
            scalar_t yT_ = ( static_cast<scalar_t>( fy    *dilation)+beta  );
            scalar_t yB_ = ( static_cast<scalar_t>((1.+fy)*dilation)-beta  );

            scalar_t xL_P = exp(SAFE_DIV(-xL_*xL_, 2*sigma*sigma));
            scalar_t xR_P = exp(SAFE_DIV(-xR_*xR_, 2*sigma*sigma));
            scalar_t yT_P = exp(SAFE_DIV(-yT_*yT_, 2*sigma*sigma));
            scalar_t yB_P = exp(SAFE_DIV(-yB_*yB_, 2*sigma*sigma));
            // if (sigma==0){
            //     printf("xL_P %.10f\n", xL_P);
            //     // printf("%.10f\n", -(xL_*xL_)/(2*sigma*sigma));

            // }

            val += static_cast<scalar_t> (yT_P*xL_P * DIM3_INDEX(input1, b, c, yT, xL));
            val += static_cast<scalar_t> (yT_P*xR_P * DIM3_INDEX(input1, b, c, yT, xR));
            val += static_cast<scalar_t> (yB_P*xL_P * DIM3_INDEX(input1, b, c, yB, xL));
            val += static_cast<scalar_t> (yB_P*xR_P * DIM3_INDEX(input1, b, c, yB, xR));
            sum += (yT_P*xL_P + yT_P*xR_P + yB_P*xL_P + yB_P*xR_P);
        }
    }    

    output[index] = SAFE_DIV(val, sum);

}


template <typename scalar_t>
__global__ void kernel_resample2d_backward_input1(
    const int n, const scalar_t* __restrict__ input1, const long4 input1_size, const long4 input1_stride,
    const scalar_t* __restrict__ input2, const long4 input2_size, const long4 input2_stride,
    const scalar_t* __restrict__ gradOutput, const long4 gradOutput_size, const long4 gradOutput_stride,
    scalar_t* __restrict__ gradInput, const long4 gradInput_size, const long4 gradInput_stride, int kernel_size, int dilation) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n) {
        return;
    }

    scalar_t sum = 0.0f;
    // scalar_t *xL_P = new scalar_t [kernel_size*kernel_size/4];
    // scalar_t *xR_P = new scalar_t [kernel_size*kernel_size/4];
    // scalar_t *yT_P = new scalar_t [kernel_size*kernel_size/4];
    // scalar_t *yB_P = new scalar_t [kernel_size*kernel_size/4];   

    int dim_b = DIM0(gradOutput_size);
    int dim_c = DIM1(gradOutput_size);
    int dim_h = DIM2(gradOutput_size);
    int dim_w = DIM3(gradOutput_size);
    int dim_chw = dim_c * dim_h * dim_w;
    int dim_hw  = dim_h * dim_w;

    int b = ( index / dim_chw ) % dim_b;
    int c = ( index / dim_hw )  % dim_c;
    int y = ( index / dim_w )   % dim_h;
    int x = ( index          )  % dim_w;

    scalar_t dx    = DIM3_INDEX(input2, b, 0, y, x);
    scalar_t dy    = DIM3_INDEX(input2, b, 1, y, x);
    scalar_t sigma = DIM3_INDEX(input2, b, 2, y, x);



    scalar_t xf = static_cast<scalar_t>(x) + dx;
    scalar_t yf = static_cast<scalar_t>(y) + dy;
    scalar_t alpha = xf - int(xf); // alpha
    scalar_t beta = yf - int(yf); // beta

    for (int fy = 0; fy < kernel_size/2; fy += 1) {
        for (int fx = 0; fx < kernel_size/2; fx += 1) {
            scalar_t xL_ = ( static_cast<scalar_t>( fx    *dilation)+alpha );
            scalar_t xR_ = ( static_cast<scalar_t>((1.+fx)*dilation)-alpha );
            scalar_t yT_ = ( static_cast<scalar_t>( fy    *dilation)+beta  );
            scalar_t yB_ = ( static_cast<scalar_t>((1.+fy)*dilation)-beta  );
            // scalar_t xL_ = ( alpha+static_cast<scalar_t>(fx) );
            // scalar_t xR_ = ( 1.-alpha+static_cast<scalar_t>(fx) );
            // scalar_t yT_ = ( beta+static_cast<scalar_t>(fy) );
            // scalar_t yB_ = ( 1-beta+static_cast<scalar_t>(fy) );

            scalar_t xL_P = exp(SAFE_DIV(-xL_*xL_, 2*sigma*sigma));
            scalar_t xR_P = exp(SAFE_DIV(-xR_*xR_, 2*sigma*sigma));
            scalar_t yT_P = exp(SAFE_DIV(-yT_*yT_, 2*sigma*sigma));
            scalar_t yB_P = exp(SAFE_DIV(-yB_*yB_, 2*sigma*sigma));
            // scalar_t xL_P = exp(SAFE_DIV(-xL_*xL_,2*sigma*sigma));
            // scalar_t xR_P = exp(-(xR_*xR_)/(2*sigma*sigma));
            // scalar_t yT_P = exp(-(yT_*yT_)/(2*sigma*sigma));
            // scalar_t yB_P = exp(-(yB_*yB_)/(2*sigma*sigma));          
            sum += (yT_P*xL_P + yT_P*xR_P + yB_P*xL_P + yB_P*xR_P);
            // printf("%f\n", SAFE_DIV(-xL_*xL_, 2*sigma*sigma));
        }
    }

    int idim_h = DIM2(input1_size);
    int idim_w = DIM3(input1_size);


    for (int fy = 0; fy < kernel_size/2; fy += 1) {
        int yT = max(min( int (floor(yf)-fy*dilation),    idim_h-1), 0);
        int yB = max(min( int (floor(yf)+(fy+1)*dilation),idim_h-1), 0);        
        // int yT = max(min( int (floor(yf)-fy  ),    idim_h-1), 0);
        // int yB = max(min( int (floor(yf)+fy+1),    idim_h-1), 0);

        for (int fx = 0; fx < kernel_size/2; fx += 1) {
            int xL = max(min( int (floor(xf)-fx*dilation  ),    idim_w-1), 0);
            int xR = max(min( int (floor(xf)+(fx+1)*dilation),  idim_w-1), 0);            
            // int xL = max(min( int (floor(xf)-fx  ),    idim_w-1), 0);
            // int xR = max(min( int (floor(xf)+fx+1),    idim_w-1), 0);

            scalar_t xL_ = ( static_cast<scalar_t>( fx    *dilation)+alpha );
            scalar_t xR_ = ( static_cast<scalar_t>((1.+fx)*dilation)-alpha );
            scalar_t yT_ = ( static_cast<scalar_t>( fy    *dilation)+beta  );
            scalar_t yB_ = ( static_cast<scalar_t>((1.+fy)*dilation)-beta  );
            // scalar_t xL_ = ( alpha+static_cast<scalar_t>(fx) );
            // scalar_t xR_ = ( 1.-alpha+static_cast<scalar_t>(fx) );
            // scalar_t yT_ = ( beta+static_cast<scalar_t>(fy) );
            // scalar_t yB_ = ( 1-beta+static_cast<scalar_t>(fy) );

            scalar_t xL_P = exp(SAFE_DIV(-xL_*xL_, 2*sigma*sigma));
            scalar_t xR_P = exp(SAFE_DIV(-xR_*xR_, 2*sigma*sigma));
            scalar_t yT_P = exp(SAFE_DIV(-yT_*yT_, 2*sigma*sigma));
            scalar_t yB_P = exp(SAFE_DIV(-yB_*yB_, 2*sigma*sigma));


            atomicAdd(&DIM3_INDEX(gradInput, b, c, (yT), (xL)), SAFE_DIV(yT_P*xL_P, sum) * DIM3_INDEX(gradOutput, b, c, y, x));
            atomicAdd(&DIM3_INDEX(gradInput, b, c, (yT), (xR)), SAFE_DIV(yT_P*xR_P, sum) * DIM3_INDEX(gradOutput, b, c, y, x));
            atomicAdd(&DIM3_INDEX(gradInput, b, c, (yB), (xL)), SAFE_DIV(yB_P*xL_P, sum) * DIM3_INDEX(gradOutput, b, c, y, x));
            atomicAdd(&DIM3_INDEX(gradInput, b, c, (yB), (xR)), SAFE_DIV(yB_P*xR_P, sum) * DIM3_INDEX(gradOutput, b, c, y, x));
        }
    }

}

template <typename scalar_t>
__global__ void kernel_resample2d_backward_input2(
    const int n, const scalar_t* __restrict__ input1, const long4 input1_size, const long4 input1_stride,
    const scalar_t* __restrict__ input2, const long4 input2_size, const long4 input2_stride,
    const scalar_t* __restrict__ gradOutput, const long4 gradOutput_size, const long4 gradOutput_stride,
    scalar_t* __restrict__ gradInput, const long4 gradInput_size, const long4 gradInput_stride, int kernel_size, int dilation) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n) {
        return;
    }

    scalar_t grad1 = 0.0;
    scalar_t grad2 = 0.0;
    scalar_t sum = 0.0;

      

    int dim_b = DIM0(gradInput_size);
    int dim_c = DIM1(gradInput_size);
    int dim_h = DIM2(gradInput_size);
    int dim_w = DIM3(gradInput_size);
    int dim_chw = dim_c * dim_h * dim_w;
    int dim_hw  = dim_h * dim_w;

    int b = ( index / dim_chw ) % dim_b;
    int c = ( index / dim_hw )  % dim_c;
    int y = ( index / dim_w )   % dim_h;
    int x = ( index          )  % dim_w;

    int odim_c = DIM1(gradOutput_size);

    scalar_t dx    = DIM3_INDEX(input2, b, 0, y, x);
    scalar_t dy    = DIM3_INDEX(input2, b, 1, y, x);
    scalar_t sigma = DIM3_INDEX(input2, b, 2, y, x);


    scalar_t xf = static_cast<scalar_t>(x) + dx;
    scalar_t yf = static_cast<scalar_t>(y) + dy;
    scalar_t alpha = xf - floor(xf); // alpha
    scalar_t beta = yf - floor(yf); // beta


    int idim_h = DIM2(input1_size);
    int idim_w = DIM3(input1_size);
    scalar_t sumgrad = 0.0;

    for (int fy = 0; fy < kernel_size/2; fy += 1) {
        int yT = max(min( int (floor(yf)-fy*dilation),    idim_h-1), 0);
        int yB = max(min( int (floor(yf)+(fy+1)*dilation),idim_h-1), 0);  

        for (int fx = 0; fx < kernel_size/2; fx += 1) {
            int xL = max(min( int (floor(xf)-fx*dilation  ),    idim_w-1), 0);
            int xR = max(min( int (floor(xf)+(fx+1)*dilation),  idim_w-1), 0);  

            scalar_t xL_ = ( static_cast<scalar_t>( fx    *dilation)+alpha );
            scalar_t xR_ = ( static_cast<scalar_t>((1.+fx)*dilation)-alpha );
            scalar_t yT_ = ( static_cast<scalar_t>( fy    *dilation)+beta  );
            scalar_t yB_ = ( static_cast<scalar_t>((1.+fy)*dilation)-beta  );

            scalar_t xL_P = exp(SAFE_DIV(-xL_*xL_, 2*sigma*sigma));
            scalar_t xR_P = exp(SAFE_DIV(-xR_*xR_, 2*sigma*sigma));
            scalar_t yT_P = exp(SAFE_DIV(-yT_*yT_, 2*sigma*sigma));
            scalar_t yB_P = exp(SAFE_DIV(-yB_*yB_, 2*sigma*sigma));             
            sum += (yT_P*xL_P + yT_P*xR_P + yB_P*xL_P + yB_P*xR_P);

            for (int ch = 0; ch < odim_c; ++ch) {
                if (c==0) {
                    grad1 += SAFE_DIV(xL_ * yT_P * xL_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yT, xL), -sigma*sigma);
                    grad1 -= SAFE_DIV(xR_ * yT_P * xR_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yT, xR), -sigma*sigma);
                    grad1 += SAFE_DIV(xL_ * yB_P * xL_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yB, xL), -sigma*sigma);
                    grad1 -= SAFE_DIV(xR_ * yB_P * xR_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yB, xR), -sigma*sigma); 
                    sumgrad += SAFE_DIV((  xL_*yT_P*xL_P - xR_*yT_P*xR_P + xL_*yB_P*xL_P - xR_*yB_P*xR_P  ), -sigma*sigma);             
                }
                else if (c==1) {
                    grad1 += SAFE_DIV(yT_ * yT_P * xL_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yT, xL), -sigma*sigma);
                    grad1 += SAFE_DIV(yT_ * yT_P * xR_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yT, xR), -sigma*sigma);
                    grad1 -= SAFE_DIV(yB_ * yB_P * xL_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yB, xL), -sigma*sigma);
                    grad1 -= SAFE_DIV(yB_ * yB_P * xR_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yB, xR), -sigma*sigma);
                    sumgrad  += SAFE_DIV(( yT_*yT_P*xL_P + yT_*yT_P*xR_P - yB_*yB_P*xL_P - yB_*yB_P*xR_P  ), -sigma*sigma);             
                }
                else if (c==2) {
                    grad1 += SAFE_DIV((yT_*yT_+xL_*xL_) * yT_P * xL_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yT, xL), sigma*sigma*sigma);                    
                    grad1 += SAFE_DIV((yT_*yT_+xR_*xR_) * yT_P * xR_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yT, xR), sigma*sigma*sigma);
                    grad1 += SAFE_DIV((yB_*yB_+xL_*xL_) * yB_P * xL_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yB, xL), sigma*sigma*sigma);
                    grad1 += SAFE_DIV((yB_*yB_+xR_*xR_) * yB_P * xR_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yB, xR), sigma*sigma*sigma);
                    sumgrad  += SAFE_DIV((  (yT_*yT_+xL_*xL_)*yT_P*xL_P + (yT_*yT_+xR_*xR_)*yT_P*xR_P + (yB_*yB_+xL_*xL_)*yB_P*xL_P + (yB_*yB_+xR_*xR_)*yB_P*xR_P  ), sigma*sigma*sigma);  

                }
            }
        }
    }

        

    for (int fy = 0; fy < kernel_size/2; fy += 1) {
        int yT = max(min( int (floor(yf)-fy*dilation),    idim_h-1), 0);
        int yB = max(min( int (floor(yf)+(fy+1)*dilation),idim_h-1), 0);  

        for (int fx = 0; fx < kernel_size/2; fx += 1) {
            int xL = max(min( int (floor(xf)-fx*dilation  ),    idim_w-1), 0);
            int xR = max(min( int (floor(xf)+(fx+1)*dilation),  idim_w-1), 0);  

            scalar_t xL_ = ( static_cast<scalar_t>( fx    *dilation)+alpha );
            scalar_t xR_ = ( static_cast<scalar_t>((1.+fx)*dilation)-alpha );
            scalar_t yT_ = ( static_cast<scalar_t>( fy    *dilation)+beta  );
            scalar_t yB_ = ( static_cast<scalar_t>((1.+fy)*dilation)-beta  );

            scalar_t xL_P = exp(SAFE_DIV(-xL_*xL_, 2*sigma*sigma));
            scalar_t xR_P = exp(SAFE_DIV(-xR_*xR_, 2*sigma*sigma));
            scalar_t yT_P = exp(SAFE_DIV(-yT_*yT_, 2*sigma*sigma));
            scalar_t yB_P = exp(SAFE_DIV(-yB_*yB_, 2*sigma*sigma));
            for (int ch = 0; ch < odim_c; ++ch) {
                grad2 += sumgrad/odim_c * yT_P * xL_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yT, xL);             
                grad2 += sumgrad/odim_c * yT_P * xR_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yT, xR);                
                grad2 += sumgrad/odim_c * yB_P * xL_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yB, xL);               
                grad2 += sumgrad/odim_c * yB_P * xR_P * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, yB, xR);   
            }
          
        }
    } 


    gradInput[index] = SAFE_DIV(grad1, sum) - SAFE_DIV(grad2, sum*sum);

}




void resample2d_kernel_forward(
    at::Tensor& input1, 
    at::Tensor& input2,
    at::Tensor& output, 
    int kernel_size,
    int dilation) {

    int n = output.numel();

    const long4 input1_size = make_long4(input1.size(0), input1.size(1), input1.size(2), input1.size(3));
    const long4 input1_stride = make_long4(input1.stride(0), input1.stride(1), input1.stride(2), input1.stride(3));

    const long4 input2_size = make_long4(input2.size(0), input2.size(1), input2.size(2), input2.size(3));
    const long4 input2_stride = make_long4(input2.stride(0), input2.stride(1), input2.stride(2), input2.stride(3));

    const long4 output_size = make_long4(output.size(0), output.size(1), output.size(2), output.size(3));
    const long4 output_stride = make_long4(output.stride(0), output.stride(1), output.stride(2), output.stride(3));

    // TODO: when atomicAdd gets resolved, change to AT_DISPATCH_FLOATING_TYPES_AND_HALF
    AT_DISPATCH_FLOATING_TYPES(input1.type(), "resample_forward_kernel", ([&] {
        kernel_resample2d_update_output<scalar_t><<< (n + CUDA_NUM_THREADS - 1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream() >>>(
            n,
            input1.data<scalar_t>(),
            input1_size,
            input1_stride, 
            input2.data<scalar_t>(),
            input2_size,
            input2_stride,
            output.data<scalar_t>(),
            output_size,
            output_stride,
            kernel_size,
            dilation);

    }));

        // TODO: ATen-equivalent check

       //    THCudaCheck(hipGetLastError());

}

void resample2d_kernel_backward(
    at::Tensor& input1,
    at::Tensor& input2,
    at::Tensor& gradOutput,
    at::Tensor& gradInput1,
    at::Tensor& gradInput2,
    int kernel_size,
    int dilation) {

    int n = gradOutput.numel();

    const long4 input1_size = make_long4(input1.size(0), input1.size(1), input1.size(2), input1.size(3));
    const long4 input1_stride = make_long4(input1.stride(0), input1.stride(1), input1.stride(2), input1.stride(3));

    const long4 input2_size = make_long4(input2.size(0), input2.size(1), input2.size(2), input2.size(3));
    const long4 input2_stride = make_long4(input2.stride(0), input2.stride(1), input2.stride(2), input2.stride(3));

    const long4 gradOutput_size = make_long4(gradOutput.size(0), gradOutput.size(1), gradOutput.size(2), gradOutput.size(3));
    const long4 gradOutput_stride = make_long4(gradOutput.stride(0), gradOutput.stride(1), gradOutput.stride(2), gradOutput.stride(3));

    const long4 gradInput1_size = make_long4(gradInput1.size(0), gradInput1.size(1), gradInput1.size(2), gradInput1.size(3));
    const long4 gradInput1_stride = make_long4(gradInput1.stride(0), gradInput1.stride(1), gradInput1.stride(2), gradInput1.stride(3));

    AT_DISPATCH_FLOATING_TYPES(input1.type(), "resample_backward_input1", ([&] {

        kernel_resample2d_backward_input1<scalar_t><<< (n + CUDA_NUM_THREADS - 1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream() >>>(
            n, 
            input1.data<scalar_t>(), 
            input1_size,
            input1_stride,
            input2.data<scalar_t>(),
            input2_size, 
            input2_stride,
            gradOutput.data<scalar_t>(),
            gradOutput_size,
            gradOutput_stride,
            gradInput1.data<scalar_t>(),
            gradInput1_size,
            gradInput1_stride, 
            kernel_size,
            dilation
        );

    }));

    const long4 gradInput2_size = make_long4(gradInput2.size(0), gradInput2.size(1), gradInput2.size(2), gradInput2.size(3));
    const long4 gradInput2_stride = make_long4(gradInput2.stride(0), gradInput2.stride(1), gradInput2.stride(2), gradInput2.stride(3));

    n = gradInput2.numel();

    AT_DISPATCH_FLOATING_TYPES(gradInput2.type(), "resample_backward_input2", ([&] {


        kernel_resample2d_backward_input2<scalar_t><<< (n + CUDA_NUM_THREADS - 1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream() >>>(
            n, 
            input1.data<scalar_t>(), 
            input1_size, 
            input1_stride,
            input2.data<scalar_t>(), 
            input2_size,
            input2_stride,
            gradOutput.data<scalar_t>(),
            gradOutput_size,
            gradOutput_stride,
            gradInput2.data<scalar_t>(),
            gradInput2_size,
            gradInput2_stride,
            kernel_size,
            dilation
       );

    }));

    // TODO: Use the ATen equivalent to get last error

    //    THCudaCheck(hipGetLastError());

}
